/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	100
#define MyType		float
								 

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    //printf("(Rowcpu)i= %d\n",i);
    for( k = -filterR; k <= filterR; k++) {
      sum += h_Src[i + k] * h_Filter[filterR - k];
      //printf("i+k=%d, filterR-k=%d\n",i+k,filterR-k);
    }
    h_Dst[i] = sum;
    //printf("sum= %f\n",sum);
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }     
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    //printf("(Colcpu)i= %d\n",i);
    for(k = -filterR; k <= filterR; k++) {
      sum += h_Src[i+k*(imageW+2*filterR)] * h_Filter[filterR-k];
      //printf("i+k*(imageW+2*filterR)=%d, filterR-k=%d\n",i+k*(imageW+2*filterR),filterR-k);
    }
    h_Dst[i] = sum;
    //printf("sum= %f\n",sum);
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }
}

__global__ void kernelRow(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;

    for(int k = -filterR; k <= filterR; k++) {
	sum += d_Input[ty * (imageW+2*filterR) + tx + k] * d_Filter[filterR - k];
    }
    d_OutputGPU[ty * (imageW+2*filterR) + tx] = sum;
}

__global__ void kernelColumn(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;

    for(int k = -filterR; k <= filterR; k++) {
	sum += d_Input[(ty+k) * (imageW+2*filterR) + tx] * d_Filter[filterR-k];
    }
    d_OutputGPU[ty * (imageW+2*filterR) + tx] = sum;
}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    MyType
    *h_Filter=0,
    *h_Input=0,
    *h_Buffer=0,
    *h_OutputCPU=0,
    *d_Filter=0,
    *d_Input=0,
    *d_Buffer=0,
    *d_OutputGPU=0;
	
    MyType MaxDiff = 0.0, element1, element2;

    int imageW;
    int imageH;
    unsigned int i,j;
    int position;
    
    hipError_t error;
    struct hipDeviceProp_t prop;
    int device;
    double timing, cpu_time, gpu_time;
    struct timeval etstart;
    struct timezone tzp;
    dim3 grid, block;    

    printf("Enter filter radius : ");
    if( scanf("%d", &filter_radius) == EOF ) {
	printf("Error reading\n");
	return(1);
    }

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.  

    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    if( scanf("%d", &imageW) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
    imageH = imageW;

    printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    // Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
    h_Filter    = (MyType *)malloc(FILTER_LENGTH * sizeof(MyType));
    h_Input     = (MyType *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(MyType));
    h_Buffer    = (MyType *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(MyType));
    h_OutputCPU = (MyType *)calloc((imageW+2*filter_radius) * (imageH+2*filter_radius), sizeof(MyType));
   
    if( h_Filter==0 || h_Input==0 || h_Buffer==0 || h_OutputCPU==0 ) {
	printf("Could not allocate memory(CPU)\n");
	if(h_Filter == 0){
	    free(h_Filter);
	}
	if(h_Input == 0){
	    free(h_Input);
	}
	if(h_Buffer == 0){
	    free(h_Buffer);
	}
	if(h_OutputCPU == 0){
	    free(h_OutputCPU);
	}
	return(1);
    }

    if( hipMalloc((void**)&d_Filter, FILTER_LENGTH * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipDeviceReset();
	return(1);
    }
    if( hipMalloc((void**)&d_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_Input, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset1Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }

    if( hipMalloc((void**)&d_Buffer, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_Buffer, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset2Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipDeviceReset();
	return(1);
    }
    if( hipMalloc((void**)&d_OutputGPU, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_OutputGPU, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset3Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	free(h_Input);
	free(h_Buffer);
	free(h_OutputCPU);
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
        hipFree(d_OutputGPU);
	hipDeviceReset();
	return(1);
    }
    // to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
    // arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
    // to convolution kai arxikopoieitai kai auth tuxaia.

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (MyType)(rand() % 16);
    }
    //printf("Init\n");
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < ((imageW+2*filter_radius)*(imageH+filter_radius));i++, j++) {
        h_Input[i] += (MyType)rand() / ((MyType)RAND_MAX / 255) + (MyType)rand() / (MyType)RAND_MAX;
	//printf("i= %d, val = %lf\n", i, h_Input[i]);
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    cpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    cpu_time = timing - cpu_time;
    printf("CPU_time is: %lf seconds\n", cpu_time);
    //for(i=0; i < (imageW+2*filter_radius)*(imageH+2*filter_radius);i++){
	//printf("i= %d, val is %lf\n",i,h_Input[i]);
    //}
    hipGetDevice(&device);
    if( hipGetDeviceProperties(&prop,device) != hipSuccess ) {
	printf("Invalid device.\n");
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    if( (imageW*imageH) > prop.maxThreadsPerBlock ) {
	block.x = sqrt(prop.maxThreadsPerBlock);
	block.y = sqrt(prop.maxThreadsPerBlock);
	grid.x = imageW/block.x;
	grid.y = imageH/block.y;
    }
    else {
	grid = 1;
    	block.x = imageW;
    	block.y = imageH;
    }
    //The code below is executed in GPU     
    printf("GPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    gpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;

    //memory copy from host to device
    if( hipMemcpy(d_Filter,h_Filter, FILTER_LENGTH * sizeof(MyType), hipMemcpyHostToDevice) != hipSuccess ) {
	printf("Problem in memeory copy\n");
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
	free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    if( hipMemcpy(d_Input,h_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType), hipMemcpyHostToDevice) != hipSuccess ) {
	printf("Problem in memeory copy\n");
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    
    kernelRow<<<grid,block>>>(d_Input,d_Filter,d_Buffer,imageW,imageH,filter_radius);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Cuda Error 1: %s\n", hipGetErrorString(error));
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    kernelColumn<<<grid,block>>>(d_Buffer,d_Filter,d_OutputGPU,imageW,imageH,filter_radius);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Cuda Error 2: %s\n", hipGetErrorString(error));
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }
    if( hipMemcpy(h_Buffer,d_OutputGPU, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType), hipMemcpyDeviceToHost) != hipSuccess ) {
	printf("Problem in memeory copy\n");
	hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        hipFree(d_Filter);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	free(h_Buffer);
	free(h_OutputCPU);
	free(h_Input);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    gpu_time = timing - gpu_time;
    printf("GPU_time is: %lf seconds\n", gpu_time);
    // Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
    // pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas  

    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
	if( ABS(h_Buffer[i]-h_OutputCPU[i]) > (MyType)MaxDiff ) {
	    MaxDiff = ABS(h_Buffer[i]-h_OutputCPU[i]);
	    element1 = h_Buffer[i];
	    element2 = h_OutputCPU[i];
	    position = i;
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }
    printf("Max difference in pos:%d-> %f ->GPU: %f, CPU: %f\n", position, MaxDiff, element1, element2);
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius); i++,j++) {
	if( ABS(h_Buffer[i]-h_OutputCPU[i]) > (MyType)accuracy ) {
	    printf("Matching failure in %d GPU: %f, CPU: %f\n",i, h_Buffer[i], h_OutputCPU[i]);
	    hipFree(d_Filter);
	    hipFree(d_Input);
	    hipFree(d_Buffer);
	    hipFree(d_OutputGPU);
	    hipDeviceReset();
            free(h_Filter);
	    free(h_Buffer);
	    free(h_OutputCPU);
	    free(h_Input);
	    return(1);
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    printf("Check completed.\n");	

    // free all the allocated memory
    free(h_OutputCPU);
    free(h_Buffer);
    free(h_Input);
    free(h_Filter);

    // Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
    hipFree(d_Filter);
    hipFree(d_Input);
    hipFree(d_Buffer);
    hipFree(d_OutputGPU);
    hipDeviceReset();   

    return 0;
}
