/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	100
#define MyType		double
								 
__constant__ MyType filter[161];
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    //printf("(Rowcpu)i= %d\n",i);
    for( k = -filterR; k <= filterR; k++) {
      sum += h_Src[i + k] * h_Filter[filterR - k];
      if(i == 19){
      	printf("cpu row=%lf*%lf, inputpos = %d, filterpos=%d\n",h_Src[i+k],h_Filter[filterR-k],i+k,filterR-k);
      }
	//printf("i+k=%d, filterR-k=%d\n",i+k,filterR-k);
    }
    h_Dst[i] = sum;
    if(i == 19)
    	printf("sum= %f\n",sum);
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }     
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    //printf("(Colcpu)i= %d\n",i);
    for(k = -filterR; k <= filterR; k++) {
      sum += h_Src[i+k*(imageW+2*filterR)] * h_Filter[filterR-k];
      if(i==7){
      	printf("cpu col=%lf*%lf, inputpos = %d, filterpos = %d\n",h_Src[i+k*(imageW+2*filterR)],h_Filter[filterR-k],i+k*(imageW+2*filterR),filterR -k);
      }
    }
    h_Dst[i] = sum;
    //printf("sum= %f\n",sum);
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }
}

__global__ void kernelRow(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;
    int input_pos = ty * (imageW+2*filterR) + tx;
    int row_length = 2*filterR + blockDim.x;
    int thread_row = threadIdx.y*row_length;
    
    
    //Array's size will be dynamically allocated at kernel launch time
    extern __shared__ MyType s_Input[];
    
    for(int k = 0; k + threadIdx.x < row_length ; k+= blockDim.x){
	s_Input[thread_row + threadIdx.x + k] = d_Input[input_pos - filterR + k];
    }

    __syncthreads();


    for(int k = -filterR; k <= filterR; k++) {
	sum += s_Input[thread_row + threadIdx.x + filterR + k] * filter[filterR - k];
    }
    d_OutputGPU[input_pos] = sum;
}

__global__ void kernelColumn(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;
    int input_pos = ty * (imageW+2*filterR) + tx;
    int col_length = blockDim.y + 2*filterR;
    int thread_row = threadIdx.y * blockDim.x;
    int input_row_len = imageW+2*filterR;
    
    //Array's size will be dynamically allocated at kernel launch time
    extern __shared__ MyType s_Input[];

    for(int k = 0; k + threadIdx.y < col_length; k+= blockDim.y ){
	s_Input[thread_row + k*blockDim.x + threadIdx.x] = d_Input[input_pos + (k - filterR)*input_row_len];
    }

    __syncthreads();

    for(int k = -filterR; k <= filterR; k++) {
	sum += s_Input[thread_row + (filterR + k)*blockDim.x + threadIdx.x] * filter[filterR-k];
    }
    d_OutputGPU[input_pos] = sum;
}

void freeAll(void *d_Input,void *d_Buffer,void *d_OutputGPU,void *h_Filter,void *h_Buffer,void *h_OutputCPU,void *h_Input) {
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipHostFree(h_Input);
}
////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    MyType
    *h_Filter=0,
    *h_Input=0,
    *h_Buffer=0,
    *h_OutputCPU=0,
    *d_Input=0,
    *d_Buffer=0,
    *d_OutputGPU=0;
	
    MyType MaxDiff = 0.0, element1, element2;

    size_t imageW, imageH;
    unsigned int i,j;
    int position;
    
    hipError_t error;
    struct hipDeviceProp_t prop;
    int device;
    double timing, cpu_time, gpu_time,memcpy_time, kernel_time, kernel_timing, memcpy_timing, memcpy_time2, memcpy_timing2, memcpy_sum;
    struct timeval etstart, mem_etstart, mem2_etstart, kernel_etstart;
    struct timezone tzp, mem_tzp, mem2_tzp, kernel_tzp;
    dim3 grid, block;    

    printf("Enter filter radius : ");
    if( scanf("%d", &filter_radius) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
    
    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    if( scanf("%zd", &imageW) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
    imageH = imageW;

    printf("Image Width x Height = %lu x %lu\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    
    h_Filter = (MyType *)malloc(FILTER_LENGTH * sizeof(MyType));
    if( h_Filter==0 ) {
	printf("Could not allocate memory(CPU)\n");
	return(1);
    }
    if( hipHostAlloc((void **)&h_Input,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)),hipHostMallocDefault) != hipSuccess) {
	printf("CudaHostAlloc error in h_Input.\n");
	return(1);
   }
    memset((void*)h_Input,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)));
    if( hipHostAlloc((void **)&h_Buffer,(size_t)((imageW+2*filter_radius) * (imageH+2*filter_radius) *(size_t)sizeof(MyType)), hipHostMallocDefault) != hipSuccess ) {
	    printf("CudaHostAlloc error in h_Buffer.\n");
	    hipHostFree(h_Input);
	    return(1);
    }
    memset((void*)h_Buffer,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType))); 
    if( hipHostAlloc((void **)&h_OutputCPU,(size_t)((imageW+2*filter_radius) * (imageH+2*filter_radius)*(size_t)sizeof(MyType)),hipHostMallocDefault) != hipSuccess) {
	    printf("CudaHpstAlloc error in h_OutputCPU.\n");
	    hipHostFree(h_Input);
	    hipHostFree(h_Buffer);
	    return(1);
    }
    memset((void*)h_OutputCPU,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)));

    if( hipMalloc((void**)&d_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_Input, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset d_Input Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }

    if( hipMalloc((void**)&d_Buffer, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_Buffer, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset d_Buffer Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipDeviceReset();
	return(1);
    }
    if( hipMalloc((void**)&d_OutputGPU, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipFree(d_Buffer);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_OutputGPU, 0, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset d_OutputGPU Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipFree(d_Buffer);
        hipFree(d_OutputGPU);
	hipDeviceReset();
	return(1);
    }

    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (MyType)(rand() % 16);
    }
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < ((imageW+2*filter_radius)*(imageH+filter_radius));i++, j++) {
        h_Input[i] += (MyType)rand() / ((MyType)RAND_MAX / 255) + (MyType)rand() / (MyType)RAND_MAX;
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    // To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
    printf("CPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }
    cpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    cpu_time = timing - cpu_time;
    printf("CPU_time is: %lf seconds\n", cpu_time);
    hipGetDevice(&device);
    if( hipGetDeviceProperties(&prop,device) != hipSuccess ) {
	printf("Invalid device.\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }
    if( (imageW*imageH) > prop.maxThreadsPerBlock ) {
	block.x = sqrt(prop.maxThreadsPerBlock);
	block.y = sqrt(prop.maxThreadsPerBlock);
	grid.x = imageW/block.x;
	grid.y = imageH/block.y;
    }
    else {
	grid = 1;
    	block.x = imageW;
    	block.y = imageH;
    }
    //The code below is executed in GPU     
    printf("GPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    gpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    
    if(gettimeofday(&mem_etstart, &mem_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    memcpy_time = ((double)mem_etstart.tv_sec) + ((double)mem_etstart.tv_usec) / 1000000.0;

    if( hipMemcpyToSymbol(HIP_SYMBOL(filter),h_Filter, FILTER_LENGTH * sizeof(MyType), 0, hipMemcpyHostToDevice) != hipSuccess ) {
	printf("Problem in memory copy\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }
    if( hipMemcpy(d_Input,h_Input, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType), hipMemcpyHostToDevice) != hipSuccess ) {
	printf("Problem in memory copy\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }
    
    if(gettimeofday(&mem_etstart, &mem_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    memcpy_timing = ((double)mem_etstart.tv_sec) + ((double)mem_etstart.tv_usec) / 1000000.0;
    memcpy_time = memcpy_timing - memcpy_time;

    if(gettimeofday(&kernel_etstart, &kernel_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    kernel_time = ((double)kernel_etstart.tv_sec) + ((double)kernel_etstart.tv_usec) / 1000000.0;

    kernelRow<<<grid,block,(block.x + 2*filter_radius)*block.y*sizeof(MyType)>>>(d_Input,filter,d_Buffer,imageW,imageH,filter_radius);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Cuda Error 1: %s\n", hipGetErrorString(error));
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    kernelColumn<<<grid,block,(block.y + 2*filter_radius)*block.x*sizeof(MyType)>>>(d_Buffer,filter,d_OutputGPU,imageW,imageH,filter_radius);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Cuda Error 2: %s\n", hipGetErrorString(error));
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    if(gettimeofday(&kernel_etstart, &kernel_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    kernel_timing = ((double)kernel_etstart.tv_sec) + ((double)kernel_etstart.tv_usec) / 1000000.0;
    kernel_time = kernel_timing - kernel_time;

    if(gettimeofday(&mem2_etstart, &mem2_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    memcpy_time2 = ((double)mem2_etstart.tv_sec) + ((double)mem2_etstart.tv_usec) / 1000000.0;

    if( hipMemcpy(h_Buffer,d_OutputGPU, (imageW+2*filter_radius) * (imageH+2*filter_radius) * sizeof(MyType), hipMemcpyDeviceToHost) != hipSuccess ) {
	printf("Problem in memeory copy\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    if(gettimeofday(&mem2_etstart, &mem2_tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    memcpy_timing2 = ((double)mem2_etstart.tv_sec) + ((double)mem2_etstart.tv_usec) / 1000000.0;
    memcpy_time2 = memcpy_timing2 - memcpy_time2;
    memcpy_sum = memcpy_time + memcpy_time2;

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
	freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    gpu_time = timing - gpu_time;
    printf("GPU_time is: %lf seconds\n", gpu_time);
    printf("Memcpy HtoD: %lf seconds\n",memcpy_time);
    printf("Memcpy DtoH: %lf seconds\n",memcpy_time2);
    printf("Kernels: %lf seconds\n",kernel_time);
    printf("Memcpysum: %lf seconds\n",memcpy_sum);
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
	if( ABS(h_Buffer[i]-h_OutputCPU[i]) > (MyType)MaxDiff ) {
	    MaxDiff = ABS(h_Buffer[i]-h_OutputCPU[i]);
	    element1 = h_Buffer[i];
	    element2 = h_OutputCPU[i];
	    position = i;
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }
    printf("Max difference in pos:%d-> %f ->GPU: %f, CPU: %f\n", position, MaxDiff, element1, element2);
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius); i++,j++) {
	if( ABS(h_Buffer[i]-h_OutputCPU[i]) > (MyType)accuracy ) {
	    printf("Matching failure in %d GPU: %f, CPU: %f\n",i, h_Buffer[i], h_OutputCPU[i]);
	    freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	    return(1);
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    printf("Check completed.\n");	
    
    //free all host and device allocated memory
    freeAll(d_Input,d_Buffer,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);

    return 0;
}
