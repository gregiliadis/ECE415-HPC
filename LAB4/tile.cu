/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	100
#define MyType		double
								 
__constant__ MyType filter[161];
////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter, 
                       int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    for( k = -filterR; k <= filterR; k++) {
      sum += h_Src[i + k] * h_Filter[filterR - k];
    }
    h_Dst[i] = sum;
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }     
}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(MyType *h_Dst, MyType *h_Src, MyType *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int k, i, j;
  
  for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
    MyType sum = 0;
    for(k = -filterR; k <= filterR; k++) {
      sum += h_Src[i+k*(imageW+2*filterR)] * h_Filter[filterR-k];
    }
    h_Dst[i] = sum;
    if(j == imageW) {
      i=i+(2*filter_radius);
	j=0;
    }
  }
}

__global__ void kernelRow(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;
    int input_pos = ty * (imageW+2*filterR) + tx;
    int row_length = 2*filterR + blockDim.x;
    int thread_row = threadIdx.y*row_length;
    
    
    //Array's size will be dynamically allocated at kernel launch time
    extern __shared__ MyType s_Input[];
    
    for(int k = 0; k + threadIdx.x < row_length ; k+= blockDim.x){
	s_Input[thread_row + threadIdx.x + k] = d_Input[input_pos - filterR + k];
    }

    __syncthreads();


    for(int k = -filterR; k <= filterR; k++) {
	sum += s_Input[thread_row + threadIdx.x + filterR + k] * filter[filterR - k];
    }
    d_OutputGPU[input_pos] = sum;
}

__global__ void kernelColumn(MyType *d_Input,MyType *d_Filter,MyType *d_OutputGPU,int imageW,int imageH,int filterR)
{
    int tx = blockIdx.x*blockDim.x+threadIdx.x + filterR;
    int ty = blockIdx.y*blockDim.y+threadIdx.y + filterR;
    MyType sum = 0;
    int input_pos = ty * (imageW+2*filterR) + tx;
    int col_length = blockDim.y + 2*filterR;
    int thread_row = threadIdx.y * blockDim.x;
    int input_row_len = imageW+2*filterR;
    
    //Array's size will be dynamically allocated at kernel launch time
    extern __shared__ MyType s_Input[];

    for(int k = 0; k + threadIdx.y < col_length; k+= blockDim.y ){
	s_Input[thread_row + k*blockDim.x + threadIdx.x] = d_Input[input_pos + (k - filterR)*input_row_len];
    }

    __syncthreads();

    for(int k = -filterR; k <= filterR; k++) {
	sum += s_Input[thread_row + (filterR + k)*blockDim.x + threadIdx.x] * filter[filterR-k];
    }
    d_OutputGPU[input_pos] = sum;
}

void free_all(void *d_Input,void *d_OutputGPU,void *h_Filter,void *h_Buffer,void *h_OutputCPU,void *h_Input) { 
	hipFree(d_Input);
	hipFree(d_OutputGPU);
	hipDeviceReset();
        free(h_Filter);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipHostFree(h_Input);
}
	

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
    
    MyType
    *h_Filter=0,
    *h_Input=0,
    *h_Buffer=0,
    *h_OutputCPU=0,
    *d_Input=0,
    *d_OutputGPU=0;
	
    MyType MaxDiff = 0.0, element1, element2;

    size_t imageW, imageH;
    unsigned int i,j;
    int position;
    int tile_size, num_of_tiles, tile_pos, tile_height;
    
    hipError_t error;
    struct hipDeviceProp_t prop;
    int device;
    double timing, cpu_time, gpu_time;
    struct timeval etstart;
    struct timezone tzp;
    dim3 grid, block;    

    printf("Enter filter radius : ");
    if( scanf("%d", &filter_radius) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
	
    printf("Enter tile size : ");
    if( scanf("%d", &tile_size) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
 
    printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
    if( scanf("%zd", &imageW) == EOF ) {
	printf("Error reading\n");
	return(1);
    }
    imageH = imageW;

    printf("Image Width x Height = %lu x %lu\n\n", imageW, imageH);
    printf("Allocating and initializing host arrays...\n");
    
    h_Filter = (MyType *)malloc(FILTER_LENGTH * sizeof(MyType));  
    if( h_Filter==0 ) {
	printf("Could not allocate memory(CPU)\n");
	return(1);
    }

    if( hipHostAlloc((void**)&h_Input,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)),hipHostMallocDefault) != hipSuccess){
	    printf("hipHostAlloc error in h_Input.\n");
	    return(1);
    }
    memset((void*)h_Input,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)));
    if( hipHostAlloc((void**)&h_Buffer,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)),hipHostMallocDefault) != hipSuccess ) {
	    printf("CudaHostAlloc error in h_Buffer.\n");
	    hipHostFree(h_Input);
	    return(1);
    }
    memset((void*)h_Buffer,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)));
    if( hipHostAlloc((void**)&h_OutputCPU,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)),hipHostMallocDefault) != hipSuccess ) {
	    printf("CudAhostAlloc error in h_OutputGPU.\n");
	    hipHostFree(h_Input);
	    hipHostFree(h_Buffer);
	    return(1);
    }
    memset((void*)h_OutputCPU,0,(size_t)((imageW+2*filter_radius)*(imageH+2*filter_radius)*(size_t)sizeof(MyType)));
    srand(200);

    for (i = 0; i < FILTER_LENGTH; i++) {
        h_Filter[i] = (MyType)(rand() % 16);
    }
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < ((imageW+2*filter_radius)*(imageH+filter_radius));i++, j++) {
        h_Input[i] += (MyType)rand() / ((MyType)RAND_MAX / 255) + (MyType)rand() / (MyType)RAND_MAX;
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    printf("CPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        free(h_Filter);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipHostFree(h_Input);
	return(1);
    }
    cpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
    convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        free(h_Filter);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipHostFree(h_Input);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    cpu_time = timing - cpu_time;
    printf("CPU_time is: %lf seconds\n", cpu_time);
    hipGetDevice(&device);
    if( hipGetDeviceProperties(&prop,device) != hipSuccess ) {
	printf("Invalid device.\n");
        free(h_Filter);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipHostFree(h_Input);
	return(1);
    }

    num_of_tiles = (imageW*imageH)/tile_size;

    if( imageW > sqrt(prop.maxThreadsPerBlock) ) {
    	    block.x = sqrt(prop.maxThreadsPerBlock);
	    grid.x = imageW/block.x;
	    if( tile_size/imageW > sqrt(prop.maxThreadsPerBlock)){
		    block.y = sqrt(prop.maxThreadsPerBlock);
		    grid.y = (tile_size/imageW)/sqrt(prop.maxThreadsPerBlock);
	    }
	    else{
	    	    block.y = tile_size/imageW;
		    grid.y = 1;
	    }
	    tile_height = tile_size/imageW;
    }
    else{
            grid=1;
	    block.x = imageW;
	    block.y = imageH;
	    num_of_tiles=1;
	    tile_height = imageH;
    }
	
    if( hipMalloc((void**)&d_Input, (imageW+2*filter_radius)*(tile_height+2*filter_radius)*sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_Input, 0, (imageW+2*filter_radius)*(tile_height+2*filter_radius)*sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset1Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }	

    if( hipMalloc((void**)&d_OutputGPU, (imageW+2*filter_radius)*(tile_height+2*filter_radius)*sizeof(MyType)) != hipSuccess ) {
	printf("Could not allocate memory(GPU)\n");
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
	hipDeviceReset();
	return(1);
    }
    /*Memset is asynchronous with respect to the host.*/
    hipMemset((void*)d_OutputGPU, 0, (imageW+2*filter_radius)*(tile_height+2*filter_radius)*sizeof(MyType));
    hipDeviceSynchronize();
    error = hipGetLastError();
    if( error != hipSuccess ) {
	printf("Memset3Error: %s\n",hipGetErrorString(error));
	free(h_Filter);
	hipHostFree(h_Input);
	hipHostFree(h_Buffer);
	hipHostFree(h_OutputCPU);
	hipFree(d_Input);
        hipFree(d_OutputGPU);
	hipDeviceReset();
	return(1);
    }
    
    //The code below is executed in GPU     
    printf("GPU computation...\n");
    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
	free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }
    gpu_time = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;

    //memory copy from host to device
    if( hipMemcpyToSymbol(HIP_SYMBOL(filter),h_Filter, FILTER_LENGTH * sizeof(MyType), 0, hipMemcpyHostToDevice) != hipSuccess ) {
	printf("Problem in memory copy\n");
	free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
	return(1);
    }

    /////////////////////////////////////////////////////////////////////////////////////////////////////////
    tile_pos = 0;
    for(i=0; i<num_of_tiles;i++){
    	if( hipMemcpy(d_Input,h_Input+tile_pos, (imageW+2*filter_radius) * (tile_height+2*filter_radius) * sizeof(MyType), hipMemcpyHostToDevice) != hipSuccess ) {
		printf("Problem in memory copy\n");
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
    	}
    	kernelRow<<<grid,block, block.y *(block.x+2*filter_radius)*sizeof(MyType)>>>(d_Input,filter,d_OutputGPU,imageW,tile_height,filter_radius);
    	//cudaDeviceSynchronize();
    	error = hipGetLastError();
    	if( error != hipSuccess ) {
		printf("Cuda Error 1: %s\n", hipGetErrorString(error));
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
   	}
    	
	if( hipMemcpy(h_Buffer+((imageW+2*filter_radius)*filter_radius)+tile_pos,
				d_OutputGPU+((imageW+2*filter_radius)*filter_radius),
			       	(imageW+2*filter_radius) * tile_height * sizeof(MyType), hipMemcpyDeviceToHost) != hipSuccess ) {
		printf("Problem in memeory copy\n");
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
    	}
    	tile_pos+= (imageW+2*filter_radius) * tile_height;
    }
    hipDeviceSynchronize();
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    tile_pos = 0;
    for(i=0; i<num_of_tiles;i++){
    	if( hipMemcpy(d_Input,h_Buffer+tile_pos, (imageW+2*filter_radius) * (tile_height+2*filter_radius) * sizeof(MyType), hipMemcpyHostToDevice) != hipSuccess ) {
		printf("Problem in memory copy\n");
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
    	}
    	kernelColumn<<<grid,block, block.x *(block.y+2*filter_radius)*sizeof(MyType)>>>(d_Input,filter,d_OutputGPU,imageW,tile_height,filter_radius);
    	//cudaDeviceSynchronize();
    	error = hipGetLastError();
    	if( error != hipSuccess ) {
		printf("Cuda Error 1: %s\n", hipGetErrorString(error));
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
   	}
    	
	if( hipMemcpy(h_Input+((imageW+2*filter_radius)*filter_radius)+tile_pos,
				d_OutputGPU+((imageW+2*filter_radius)*filter_radius),
			       	(imageW+2*filter_radius) * tile_height * sizeof(MyType), hipMemcpyDeviceToHost) != hipSuccess ) {
		printf("Problem in memeory copy\n");
		free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_OutputCPU,h_Input);
		return(1);
    	}
    	tile_pos+= (imageW+2*filter_radius) * tile_height;
    }

    if(gettimeofday(&etstart, &tzp) == -1){
        perror("Error calling gettimeofday()\n");
        free_all(d_Input, d_OutputGPU,h_Filter, h_Input, h_Buffer, h_OutputCPU);
	return(1);
    }

    timing = ((double)etstart.tv_sec) + ((double)etstart.tv_usec) / 1000000.0;
    gpu_time = timing - gpu_time;
    printf("GPU_time is: %lf seconds\n", gpu_time);
    
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius);i++, j++) {
	if( ABS(h_Input[i]-h_OutputCPU[i]) > (MyType)MaxDiff ) {
	    MaxDiff = ABS(h_Input[i]-h_OutputCPU[i]);
	    element1 = h_Input[i];
	    element2 = h_OutputCPU[i];
	    position = i;
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }
    printf("Max difference in pos:%d-> %f ->GPU: %f, CPU: %f\n", position, MaxDiff, element1, element2);
    for (i =(imageW+2*filter_radius)*filter_radius+filter_radius, j = 1; i < (imageW+2*filter_radius)*(imageH+filter_radius); i++,j++) {
	if( ABS(h_Input[i]-h_OutputCPU[i]) > (MyType)accuracy ) {
	    printf("Matching failure in %d GPU: %f, CPU: %f\n",i, h_Input[i], h_OutputCPU[i]);
	    free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_Input,h_OutputCPU);
	    return(1);
	}
	if(j == imageW) {
	    i=i+(2*filter_radius);
	    j=0;
	}
    }

    printf("Check completed.\n");	

    //free all host and device allocated memory
    free_all(d_Input,d_OutputGPU,h_Filter,h_Buffer,h_Input,h_OutputCPU);

    return 0;
}
